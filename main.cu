
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

__global__ void add_vector(float *c, const float *a, const float *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

//for an array of size, fill it with a random float in [0,1]
void random_floats(float *A, int size)
{
	for (int i = 0; i < size; i++)
	{
		A[i] = ((float) rand()) / ((float) RAND_MAX);
	}
}


float my_abs(float a)
{
	if (a< 0)
		return -1 * a;
	return a;
}

//finds the machine epsilon for a float
float find_eps()
{
	float machEps = (float) 1.0;

        do {
           machEps /= (float) 2.0;
        }
        while ((float)(1.0 + machEps) != 1.0);

        return machEps;
}

int main()
{
	srand(time(0));
	const int SIZE = 512;
	size_t bytes = 512 * sizeof(float);

	//initialize out pointers on host and device
	float *A, *B, *C;
	float *dA, *dB, *dC;

	//allocate vectors on host
	A = (float*)malloc(bytes);
	random_floats(A, SIZE);
	B = (float*)malloc(bytes);
	random_floats(B, SIZE);
	C = (float*)malloc(bytes);

	//alocate vectors on device
	hipMalloc((void**)&dA, bytes);
	hipMalloc((void**)&dB, bytes);
	hipMalloc((void**)&dC, bytes);

	//copy the vector from host to device
	hipMemcpy(dA, A, bytes, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, bytes, hipMemcpyHostToDevice);

	//perform the addition
	add_vector<<<1, SIZE >>>(dC, dB, dA);

	//copy our answer back to the cpu
	hipMemcpy(C, dC, bytes, hipMemcpyDeviceToHost);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);

	//check for correctness
	float eps = find_eps();
	
	int worked = 0;
	for (int i = 0; i < SIZE; i++)
	{
		float rel_error = my_abs( ( (A[i] + B[i]) - C[i]) /C[i]);

		if ( rel_error > eps)
		{
			printf("messed up on index %d\n", i);
			printf("Calculation did not work\n");
			worked = 1;
			break;
		}
	}

	if (worked == 0)
		printf("Congrats, everyting worked!\n");
	

	//free up the host memory
	free(A);
	free(B);
	free(C);

}
